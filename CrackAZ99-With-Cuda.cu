#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/***********************************************************************
*******
  Demonstrates how to crack an encrypted password using a simple
  "brute force" algorithm. Works on passwords that consist only of 2
uppercase
  letters and a 2 digit integer. Your personalised data set is included
in the
  code. 


  Compile with:
    nvcc -o CrackAZ99-With-Cuda CrackAZ99-With-Cuda.cu 

  If you want to analyse the results then use the redirection operator
to send
  output to a file that you can view using an editor or the less
utility:

    ./CrackAZ99-With-Cuda > results.txt

  Dr Kevan Buckley, University of Wolverhampton, 2018
************************************************************************
******/
__device__ int is_a_match(char *attempts) {	// Compares each password attempt.
  char plain_passwords1[] = "IT8312";
  char plain_passwords2[] = "RB3211";
  char plain_passwords3[] = "AV7213";
  char plain_passwords4[] = "ES2114";

  char *x = attempts;
  char *p = plain_passwords1;
  char *q = plain_passwords2;
  char *r = plain_passwords3;
  char *s = plain_passwords4;


  while(*x == *p) {
    if(*x== '\0') {
  printf("Found password successfully: %s\n",plain_passwords1);
      break;
    }
    x++;
    p++;
  }
  
while(*x == *q) {
    if(*x== '\0') {
  printf("Found password successfully: %s\n",plain_passwords2);
      break;
    }
    x++;
    q++;
  }
while(*x == *r) {
    if(*x== '\0') {
  
 printf("Found password successfully: %s\n",plain_passwords3); 
    break;
    }
    x++;
    r++;
  }
while(*x == *s) {
    if(*x== '\0') {
  printf("Found password successfully: %s\n",plain_passwords4);
      return 1;
    }
    x++;
    s++;
  }

  return 0;
}


__global__ void  kernel() {
char k,l,m,n;
  
  char password[7];
  password[6] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i; 
char secondValue = j; 
    
password[0] = firstValue;
password[1] = secondValue;
	for(k='0'; k<='9'; k++){
	  for(l='0'; l<='9'; l++){
	   for(m='0'; m<='9'; m++){
	     for(n='0'; n<='9'; n++){
	        password[2] = k;
	        password[3] = l;
	        password[4] = m;
	        password[5] = n; 
	      if(is_a_match(password)) {
		//printf("Success");
	      } 
             else {
	     //printf("tried: %s\n", password);		  
	         }
	      }
	   }
	}
    }
}

int time_difference(struct timespec *start, 
                    struct timespec *finish, 
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}

int main() {
// starting kernel
  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

  kernel <<<26, 26>>>();

  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

  return 0;
}
